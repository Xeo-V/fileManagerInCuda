#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <unordered_map>
#include <list>
#include <vector>

std::unordered_map<std::string, std::string> fileSystem;
std::unordered_map<std::string, std::list<std::string>::iterator> cacheMap;
std::list<std::string> lruList;

__global__ void stringSearchKernel(const char* text, const char* query, int* positions, int textSize, int querySize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx + querySize > textSize) return;

    bool match = true;
    for (int i = 0; i < querySize; ++i) {
        if (text[idx + i] != query[i]) {
            match = false;
            break;
        }
    }
    positions[idx] = (match) ? 1 : 0;
}

void initializeFileSystem() {
    fileSystem.clear();
    cacheMap.clear();
    lruList.clear();
}

bool createFile(const std::string& fileName) {
    if (fileSystem.find(fileName) != fileSystem.end()) {
        return false;
    }
    fileSystem[fileName] = "";
    return true;
}

bool writeToFile(const std::string& fileName, const std::string& data) {
    if (fileSystem.find(fileName) == fileSystem.end()) {
        return false;
    }
    fileSystem[fileName] = data;
    return true;
}

bool readFromFile(const std::string& fileName, std::string& data) {
    if (fileSystem.find(fileName) == fileSystem.end()) {
        return false;
    }

    if (cacheMap.find(fileName) != cacheMap.end()) {
        lruList.erase(cacheMap[fileName]);
        lruList.push_front(fileName);
        cacheMap[fileName] = lruList.begin();
        data = fileSystem[fileName];
    }
    else {
        if (lruList.size() == 5) {
            std::string last = lruList.back();
            lruList.pop_back();
            cacheMap.erase(last);
        }

        lruList.push_front(fileName);
        cacheMap[fileName] = lruList.begin();
        data = fileSystem[fileName];
    }

    return true;
}

bool searchStringInFile(const std::string& fileName, const std::string& query, std::vector<int>& positions) {
    std::string fileContent;
    if (!readFromFile(fileName, fileContent)) {
        return false;
    }

    int* d_positions;
    char* d_text, * d_query;
    int textSize = fileContent.size();
    int querySize = query.size();
    positions.resize(textSize, 0);

    hipMalloc(&d_positions, textSize * sizeof(int));
    hipMalloc(&d_text, textSize * sizeof(char));
    hipMalloc(&d_query, querySize * sizeof(char));

    hipMemcpy(d_text, fileContent.c_str(), textSize * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_query, query.c_str(), querySize * sizeof(char), hipMemcpyHostToDevice);

    stringSearchKernel << <(textSize + 255) / 256, 256 >> > (d_text, d_query, d_positions, textSize, querySize);

    hipMemcpy(positions.data(), d_positions, textSize * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_positions);
    hipFree(d_text);
    hipFree(d_query);

    return true;
}

int main() {
    initializeFileSystem();

    while (true) {
        std::cout << "\nChoose an operation:\n";
        std::cout << "1. Create file\n";
        std::cout << "2. Write to file\n";
        std::cout << "3. Read from file\n";
        std::cout << "4. List files\n";
        std::cout << "5. Search string in file\n";
        std::cout << "6. Exit\n";

        int choice;
        std::cin >> choice;

        std::string fileName, data, query;
        std::vector<int> positions;
        switch (choice) {
        case 1:
            std::cout << "Enter the file name: ";
            std::cin >> fileName;
            createFile(fileName);
            break;
        case 2:
            std::cout << "Enter the file name: ";
            std::cin >> fileName;
            std::cout << "Enter the data: ";
            std::cin.ignore();
            std::getline(std::cin, data);
            writeToFile(fileName, data);
            break;
        case 3:
            std::cout << "Enter the file name: ";
            std::cin >> fileName;
            if (readFromFile(fileName, data)) {
                std::cout << "Data in " << fileName << ": " << data << '\n';
            }
            break;
        case 4:
            for (const auto& entry : fileSystem) {
                std::cout << entry.first << '\n';
            }
            break;
        case 5:
            std::cout << "Enter the file name: ";
            std::cin >> fileName;
            std::cout << "Enter the search query: ";
            std::cin >> query;
            if (searchStringInFile(fileName, query, positions)) {
                std::cout << "Positions found: ";
                for (int i = 0; i < positions.size(); ++i) {
                    if (positions[i] == 1) {
                        std::cout << i << ' ';
                    }
                }
                std::cout << '\n';
            }
            break;
        case 6:
            return 0;
        default:
            std::cout << "Invalid choice!\n";
            break;
        }
    }

    return 0;
}
